#include "hip/hip_runtime.h"
#include "postprocess.h"


__global__ void decode_kernel(float* dst, float* src, int num_detections, int detection_attribute_size, float conf_threshold) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= num_detections) return;

    // 按列优先访问数据
    float* ptr = src + idx; // 每列的第 idx 个检测结果
    float* ptr_conf = src + num_detections * 4 + idx; // 置信度部分

    float* ptr_dst = dst + idx * 7;

    // 提取边界框坐标
    const float cx = ptr[0 * num_detections];
    const float cy = ptr[1 * num_detections];
    const float ow = ptr[2 * num_detections];
    const float oh = ptr[3 * num_detections];

    // 查找最大置信度及其类别
    float max_conf = 0.0f;
    int cls_id = -1;
    for (int i = 0; i < detection_attribute_size - 4; ++i) {
        float conf = ptr_conf[i * num_detections];
        if (conf > max_conf) {
            max_conf = conf;
            cls_id = i; // 索引从 0 开始
        }
    }

    // 如果最大置信度大于阈值，则存储结果
    if (max_conf >= conf_threshold) {
        ptr_dst[0] = cx;
        ptr_dst[1] = cy;
        ptr_dst[2] = ow;
        ptr_dst[3] = oh;
        ptr_dst[4] = max_conf;
        ptr_dst[5] = cls_id;
        ptr_dst[6] = 1; // 标记为有效检测
    }
}

static __device__ float box_iou(float ax, float ay, float aw, float ah, float bx, float by,
    float bw, float bh) {
float aleft = ax - aw / 2.0f;
float atop = ay - ah / 2.0f;
float aright = ax + aw / 2.0f;
float abottom = ay + ah / 2.0f;
float bleft = bx - bw / 2.0f;
float btop = by - bh / 2.0f;
float bright = bx + bw / 2.0f;
float bbottom = by + bh / 2.0f;
float cleft = max(aleft, bleft);
float ctop = max(atop, btop);
float cright = min(aright, bright);
float cbottom = min(abottom, bbottom);
float c_area = max(cright - cleft, 0.0f) * max(cbottom - ctop, 0.0f);
if (c_area == 0.0f)
return 0.0f;

float a_area = max(0.0f, aright - aleft) * max(0.0f, abottom - atop);
float b_area = max(0.0f, bright - bleft) * max(0.0f, bbottom - btop);
return c_area / (a_area + b_area - c_area);
}

static __global__ void nms_kernel(float* bboxes, int max_objects, float threshold) {
    int position = (blockDim.x * blockIdx.x + threadIdx.x);
    if (position >= max_objects)
        return;

    float* pcurrent = bboxes  + position * 7;
    for (int i = 0; i < max_objects; ++i) {
        float* pitem = bboxes + i * 7;
        if (i == position || pcurrent[5] != pitem[5])
            continue;
        if (pitem[4] >= pcurrent[4]) {
            if (pitem[4] == pcurrent[4] && i < position)
                continue;
            float iou =
                    box_iou(pcurrent[0], pcurrent[1], pcurrent[2], pcurrent[3], pitem[0], pitem[1], pitem[2], pitem[3]);
            if (iou > threshold) {
                atomicExch(&pcurrent[6], 0);
                return;
            }
        }
    }
}

void cuda_decode(float* src,float* dst, vector<Rect>& boxes, vector<int>& class_ids, vector<float>& confidences,
    int detection_attribute_size,int num_detections, float conf_threshold, float nms_threshold,hipStream_t stream){

    int block = 256;
    int grid = ceil(num_detections / (float)block);
    float *dst_cpu = (new float[num_detections * 7]);

    CUDA_CHECK(hipMemset(dst, 0, num_detections * 7 * sizeof(float)));
    
    decode_kernel<<<grid, block, 0, stream>>>(dst,src,num_detections,detection_attribute_size,conf_threshold);
    nms_kernel<<<grid, block, 0, stream>>>(dst, num_detections, nms_threshold);
    CUDA_CHECK(hipMemcpyAsync(dst_cpu, dst, num_detections * 7 * sizeof(float), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));
    for (int i = 0; i < num_detections; ++i) {
        float* ptr = dst_cpu + i * 7;
        if (ptr[6] == 1) {
            Rect box;
            const float cx = ptr[0];
            const float cy = ptr[1];
            const float ow = ptr[2];
            const float oh = ptr[3];
            const float score = ptr[4];
            const int class_id = static_cast<int>(ptr[5]);
            // Calculate top-left corner of the bounding box
            box.x = static_cast<int>((cx - 0.5 * ow));
            box.y = static_cast<int>((cy - 0.5 * oh));
            // Set width and height of the bounding box
            box.width = static_cast<int>(ow);
            box.height = static_cast<int>(oh);

            // Store the bounding box, class ID, and confidence
            boxes.push_back(box);
            class_ids.push_back(class_id);
            confidences.push_back(score);
        }
    }
        delete[] dst_cpu;
    }